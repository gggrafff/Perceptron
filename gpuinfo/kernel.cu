
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

int main()
{
	int deviceCount;
	hipDeviceProp_t devProp;
	hipGetDeviceCount(&deviceCount);

	printf("Found %d devices\n", deviceCount);

	for (int device=0; device < deviceCount; device++)
	{
		hipGetDeviceProperties(&devProp, device);

		printf("Device %d\n", device);
		printf("Compute capability: %d.%d\n", devProp.major, devProp.minor);
		printf("Name: %s\n", devProp.name);
		printf("Total Global Memory: %d\n", devProp.totalGlobalMem);
		printf("Shared memory per block: %d\n", devProp.sharedMemPerBlock);
		printf("Registers per block: %d\n", devProp.regsPerBlock);
		printf("Warp size: %d\n", devProp.warpSize);
		printf("Max threads per block: %d\n", devProp.maxThreadsPerBlock);
		printf("Total constant memory: %d\n", devProp.totalConstMem);
		printf("Clock Rate: %d\n", devProp.clockRate);
		printf("Texture Alignment: %u\n", devProp.textureAlignment);
		printf("Device overlap: %d\n", devProp.deviceOverlap);
		printf("Multiprocessot Count: %d\n", devProp.multiProcessorCount);
		printf("Max Threads Dim: %d %d %d\n", devProp.maxThreadsDim[0], devProp.maxThreadsDim[1], devProp.maxThreadsDim[2]);
		printf("Max Grid Size: %d %d %d\n", devProp.maxGridSize[0], devProp.maxGridSize[1], devProp.maxGridSize[2]);
	}
	getchar();
    return 0;
}
